#include "hip/hip_runtime.h"
#include "CudaSimulation.cuh"

#include <functional>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <>
#include <cuda/hip/hip_runtime_api.h>

#include "Base/Exceptions.h"
#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/GpuSettings.h"

#include "Base/LoggingService.h"
#include "Base/ServiceLocator.h"
#include "AccessKernels.cuh"
#include "AccessTOs.cuh"
#include "Base.cuh"
#include "CleanupKernels.cuh"
#include "ConstantMemory.cuh"
#include "CudaMemoryManager.cuh"
#include "CudaMonitorData.cuh"
#include "Entities.cuh"
#include "Map.cuh"
#include "MonitorKernels.cuh"
#include "ActionKernels.cuh"
#include "RenderingKernels.cuh"
#include "SimulationData.cuh"
#include "SimulationKernels.cuh"
#include "SimulationResult.cuh"
#include "SelectionResult.cuh"
#include "RenderingData.cuh"

namespace
{
    class CudaInitializer
    {
    public:
        static void init() { [[maybe_unused]] static CudaInitializer instance; }

        CudaInitializer()
        {
            int deviceNumber = getDeviceNumberOfHighestComputeCapability();

            auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
            auto result = hipSetDevice(deviceNumber);
            if (result != hipSuccess) {
                throw SystemRequirementNotMetException("CUDA device could not be initialized.");
            }

            std::stringstream stream;
            stream << "device " << deviceNumber << " is set";
            loggingService->logMessage(Priority::Important, stream.str());
        }

        ~CudaInitializer() { hipDeviceReset(); }

    private:
        int getDeviceNumberOfHighestComputeCapability()
        {
            auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
            int result = 0;
            int numberOfDevices;
            CHECK_FOR_CUDA_ERROR(hipGetDeviceCount(&numberOfDevices));
            if (numberOfDevices < 1) {
                throw SystemRequirementNotMetException("No CUDA device found.");
            }
            {
                std::stringstream stream;
                if (1 == numberOfDevices) {
                    stream << "1 CUDA device found";
                } else {
                    stream << numberOfDevices << " CUDA devices found";
                }
                loggingService->logMessage(Priority::Important, stream.str());
            }

            int highestComputeCapability = 0;
            for (int deviceNumber = 0; deviceNumber < numberOfDevices; ++deviceNumber) {
                hipDeviceProp_t prop;
                CHECK_FOR_CUDA_ERROR(hipGetDeviceProperties(&prop, deviceNumber));

                std::stringstream stream;
                stream << "device " << deviceNumber << ": " << prop.name << " with compute capability " << prop.major
                       << "." << prop.minor;
                loggingService->logMessage(Priority::Important, stream.str());

                int computeCapability = prop.major * 100 + prop.minor;
                if (computeCapability > highestComputeCapability) {
                    result = deviceNumber;
                    highestComputeCapability = computeCapability;
                }
            }
            if (highestComputeCapability < 600) {
                throw SystemRequirementNotMetException(
                    "No CUDA device with compute capability of 6.0 or higher found.");
            }

            return result;
        }
    };
}

void _CudaSimulation::initCuda()
{
    CudaInitializer::init();
}

_CudaSimulation::_CudaSimulation(uint64_t timestep, Settings const& settings, GpuSettings const& gpuSettings)
{
    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    setSimulationParameters(settings.simulationParameters);
    setSimulationParametersSpots(settings.simulationParametersSpots);
    setGpuConstants(gpuSettings);
    setFlowFieldSettings(settings.flowFieldSettings);

    auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
    loggingService->logMessage(Priority::Important, "initialize simulation");

    _currentTimestep.store(timestep);
    _cudaSimulationData = new SimulationData();
    _cudaRenderingData = new RenderingData();
    _cudaSimulationResult = new SimulationResult();
    _cudaSelectionResult = new SelectionResult();
    _cudaAccessTO = new DataAccessTO();
    _cudaMonitorData = new CudaMonitorData();

    int2 worldSize{settings.generalSettings.worldSizeX, settings.generalSettings.worldSizeY};
    _cudaSimulationData->init(worldSize);
    _cudaRenderingData->init();
    _cudaMonitorData->init();
    _cudaSimulationResult->init();
    _cudaSelectionResult->init();

    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numStringBytes);
    CudaMemoryManager::getInstance().acquireMemory<char>(Const::MetadataMemorySize, _cudaAccessTO->stringBytes);

    //default array sizes for empty simulation (will be resized later if not sufficient)
    resizeArrays({100000, 100000, 10000});
}

_CudaSimulation::~_CudaSimulation()
{
    _cudaSimulationData->free();
    _cudaRenderingData->free();
    _cudaMonitorData->free();
    _cudaSimulationResult->free();
    _cudaSelectionResult->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->stringBytes);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numStringBytes);

    auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
    loggingService->logMessage(Priority::Important, "close simulation");

    delete _cudaAccessTO;
    delete _cudaSimulationData;
    delete _cudaRenderingData;
    delete _cudaMonitorData;
}

void* _CudaSimulation::registerImageResource(GLuint image)
{
    hipGraphicsResource* cudaResource;

    CHECK_FOR_CUDA_ERROR(
        hipGraphicsGLRegisterImage(&cudaResource, image, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));

    return reinterpret_cast<void*>(cudaResource);
}

void _CudaSimulation::calcCudaTimestep()
{
    KERNEL_CALL_HOST(calcSimulationTimestepKernel, *_cudaSimulationData, *_cudaSimulationResult);
    automaticResizeArrays();
    ++_currentTimestep;
}

void _CudaSimulation::drawVectorGraphics(
    float2 const& rectUpperLeft,
    float2 const& rectLowerRight,
    void* cudaResource,
    int2 const& imageSize,
    double zoom)
{
    auto cudaResourceImpl = reinterpret_cast<hipGraphicsResource*>(cudaResource);
    CHECK_FOR_CUDA_ERROR(hipGraphicsMapResources(1, &cudaResourceImpl));

    hipArray* mappedArray;
    CHECK_FOR_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&mappedArray, cudaResourceImpl, 0, 0));

    _cudaRenderingData->resizeImageIfNecessary(imageSize);

    KERNEL_CALL_HOST(
        drawImageKernel,
        rectUpperLeft,
        rectLowerRight,
        imageSize,
        static_cast<float>(zoom),
        *_cudaSimulationData,
        *_cudaRenderingData);

    const size_t widthBytes = sizeof(uint64_t) * imageSize.x;
    CHECK_FOR_CUDA_ERROR(hipMemcpy2DToArray(
        mappedArray,
        0,
        0,
        _cudaRenderingData->imageData,
        widthBytes,
        widthBytes,
        imageSize.y,
        hipMemcpyDeviceToDevice));

    CHECK_FOR_CUDA_ERROR(hipGraphicsUnmapResources(1, &cudaResourceImpl));
}

void _CudaSimulation::getSimulationData(
    int2 const& rectUpperLeft,
    int2 const& rectLowerRight,
    DataAccessTO const& dataTO)
{
    KERNEL_CALL_HOST(
        cudaGetSimulationAccessDataKernel, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);

    CHECK_FOR_CUDA_ERROR(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(dataTO.numStringBytes, _cudaAccessTO->numStringBytes, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.particles,
        _cudaAccessTO->particles,
        sizeof(ParticleAccessTO) * (*dataTO.numParticles),
        hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.stringBytes,
        _cudaAccessTO->stringBytes,
        sizeof(char) * (*dataTO.numStringBytes),
        hipMemcpyDeviceToHost));
}

void _CudaSimulation::getOverlayData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    KERNEL_CALL_HOST(
        cudaGetSimulationOverlayDataKernel, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);
    CHECK_FOR_CUDA_ERROR(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
}

void _CudaSimulation::setSimulationData(DataAccessTO const& dataTO)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(
        hipMemcpy(_cudaAccessTO->numStringBytes, dataTO.numStringBytes, sizeof(int), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->particles,
        dataTO.particles,
        sizeof(ParticleAccessTO) * (*dataTO.numParticles),
        hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));
    CHECK_FOR_CUDA_ERROR(hipMemcpy(
        _cudaAccessTO->stringBytes,
        dataTO.stringBytes,
        sizeof(char) * (*dataTO.numStringBytes),
        hipMemcpyHostToDevice));

    KERNEL_CALL_HOST(cudaSetSimulationAccessDataKernel, *_cudaSimulationData, *_cudaAccessTO);
}

void _CudaSimulation::applyForce(ApplyForceData const& applyData)
{
    KERNEL_CALL_HOST(cudaApplyForce, applyData, *_cudaSimulationData);
}

void _CudaSimulation::switchSelection(SwitchSelectionData const& switchData)
{
    KERNEL_CALL_HOST(cudaSwitchSelection, switchData, *_cudaSimulationData);
}

void _CudaSimulation::setSelection(SetSelectionData const& selectionData)
{
    KERNEL_CALL_HOST(cudaSetSelection, selectionData, *_cudaSimulationData);
}

 SelectionShallowData _CudaSimulation::getSelectionShallowData()
{
     KERNEL_CALL_HOST(cudaGetSelectionShallowData, *_cudaSimulationData, *_cudaSelectionResult);
    return _cudaSelectionResult->getSelectionShallowData();
 }

void _CudaSimulation::shallowUpdateSelection(ShallowUpdateSelectionData const& shallowUpdateData)
{
    KERNEL_CALL_HOST(cudaShallowUpdateSelection, shallowUpdateData, *_cudaSimulationData);
}

void _CudaSimulation::removeSelection()
{
    KERNEL_CALL_HOST(cudaRemoveSelection, *_cudaSimulationData);
}

void _CudaSimulation::setGpuConstants(GpuSettings const& gpuConstants_)
{
    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(gpuConstants), &gpuConstants_, sizeof(GpuSettings), 0, hipMemcpyHostToDevice));
}

auto _CudaSimulation::getArraySizes() const -> ArraySizes
{
    return {
        _cudaSimulationData->entities.cells.getSize_host(),
        _cudaSimulationData->entities.particles.getSize_host(),
        _cudaSimulationData->entities.tokens.getSize_host()};
}

OverallStatistics _CudaSimulation::getMonitorData()
{
    KERNEL_CALL_HOST(cudaGetCudaMonitorData, *_cudaSimulationData, *_cudaMonitorData);
    
    OverallStatistics result;

    auto monitorData = _cudaMonitorData->getMonitorData(getCurrentTimestep());
    result.timeStep = monitorData.timeStep;
    result.numCells = monitorData.numCells;
    result.numParticles = monitorData.numParticles;
    result.numTokens = monitorData.numTokens;
    result.totalInternalEnergy = monitorData.totalInternalEnergy;

    auto processStatistics = _cudaSimulationResult->getStatistics();
    result.numCreatedCells = processStatistics.createdCells;
    result.numSuccessfulAttacks = processStatistics.sucessfulAttacks;
    result.numFailedAttacks = processStatistics.failedAttacks;
    result.numMuscleActivities = processStatistics.muscleActivities;
    return result;
}

uint64_t _CudaSimulation::getCurrentTimestep() const
{
    return _currentTimestep.load();
}

void _CudaSimulation::setCurrentTimestep(uint64_t timestep)
{
    _currentTimestep.store(timestep);
}

void _CudaSimulation::setSimulationParameters(SimulationParameters const& parameters)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaSimulationParameters), &parameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
}

void _CudaSimulation::setSimulationParametersSpots(SimulationParametersSpots const& spots)
{
    CHECK_FOR_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaSimulationParametersSpots), &spots, sizeof(SimulationParametersSpots), 0, hipMemcpyHostToDevice));
}

void _CudaSimulation::setFlowFieldSettings(FlowFieldSettings const& settings)
{
    CHECK_FOR_CUDA_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(cudaFlowFieldSettings), &settings, sizeof(FlowFieldSettings), 0, hipMemcpyHostToDevice));
}


void _CudaSimulation::clear()
{
    KERNEL_CALL_HOST(cudaClearData, *_cudaSimulationData);
}

void _CudaSimulation::resizeArraysIfNecessary(ArraySizes const& additionals)
{
    if (_cudaSimulationData->shouldResize(
            additionals.cellArraySize, additionals.particleArraySize, additionals.tokenArraySize)) {
        resizeArrays(additionals);
    }
}

void _CudaSimulation::automaticResizeArrays()
{
    //make check after every 10th time step
    if (_currentTimestep.load() % 10 == 0) {
        if (_cudaSimulationResult->isArrayResizeNeeded()) {
            resizeArrays({0, 0, 0});
        }
    }
}

void _CudaSimulation::resizeArrays(ArraySizes const& additionals)
{
    auto loggingService = ServiceLocator::getInstance().getService<LoggingService>();
    loggingService->logMessage(Priority::Important, "resize arrays");

    _cudaSimulationData->resizeEntitiesForCleanup(
        additionals.cellArraySize, additionals.particleArraySize, additionals.tokenArraySize);
    if (!_cudaSimulationData->isEmpty()) {
        KERNEL_CALL_HOST(cudaCopyEntities, *_cudaSimulationData);
        _cudaSimulationData->resizeRemainings();
        _cudaSimulationData->swap();
    } else {
        _cudaSimulationData->resizeRemainings();
    }

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);

    auto cellArraySize = _cudaSimulationData->entities.cells.getSize_host();
    auto tokenArraySize = _cudaSimulationData->entities.tokens.getSize_host();
    CudaMemoryManager::getInstance().acquireMemory<CellAccessTO>(cellArraySize, _cudaAccessTO->cells);
    CudaMemoryManager::getInstance().acquireMemory<ParticleAccessTO>(cellArraySize, _cudaAccessTO->particles);
    CudaMemoryManager::getInstance().acquireMemory<TokenAccessTO>(tokenArraySize, _cudaAccessTO->tokens);

    CHECK_FOR_CUDA_ERROR(hipGetLastError());

    loggingService->logMessage(Priority::Unimportant, "cell array size: " + std::to_string(cellArraySize));
    loggingService->logMessage(Priority::Unimportant, "particle array size: " + std::to_string(cellArraySize));
    loggingService->logMessage(Priority::Unimportant, "token array size: " + std::to_string(tokenArraySize));

        auto const memorySizeAfter = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();
    loggingService->logMessage(Priority::Important, std::to_string(memorySizeAfter / (1024 * 1024)) + " MB GPU memory acquired");
}
